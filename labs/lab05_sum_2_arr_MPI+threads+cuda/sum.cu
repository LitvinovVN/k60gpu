#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <thread>
#include <vector>
#include "sum.h"



__global__ void printHelloFromThreadN_kernel(int n){
	printf("hello from thread %d\n", n);	
}
  
void thread_func(int n){
	hipSetDevice(n);
	printHelloFromThreadN_kernel<<<1,1>>>(n);
	hipDeviceSynchronize();
}




void thread_sum(double* a, double* b, double* c_par, size_t nStart, size_t numElementsPerThread) {
    for(int indx = nStart; indx < nStart+numElementsPerThread; indx++)
	{
		c_par[indx] = a[indx] + b[indx];
	}
}

void sum2Arrays(double* a, double* b, double* c_par, size_t cpuThreadsPerNode, size_t numElementsPerThread,
    size_t numGpu, size_t numElementsPerGpu){
    std::vector<std::thread> threads;
	for(int i = 0; i < cpuThreadsPerNode; i++) {
		size_t nStart = i * numElementsPerThread;		
		std::thread thr(thread_sum, a, b, c_par, nStart, numElementsPerThread);
		threads.emplace_back(std::move(thr));
	}
	
    /////
    int n = 0;
  	hipError_t err = hipGetDeviceCount(&n);
  	if (err != hipSuccess) {std::cout << "error " << (int)err << std::endl; return;}

  	std::vector<std::thread> t;
  	for (int i = 0; i < n; i++)
    	t.push_back(std::thread(thread_func, i));
  	std::cout << n << " threads started" << std::endl;

  	for (int i = 0; i < n; i++)
    	t[i].join();
  	std::cout << "join finished" << std::endl;
    /////

	for(auto& thr : threads) {
		thr.join();
	}    
}



extern "C"
void testSum2Arrays(int mpi_rank, int mpi_size,
    size_t cpuThreadsPerNode, size_t numElementsPerThread,
    size_t numGpu, size_t numElementsPerGpu)
{
    std::cout << "----------------------------------------" << std::endl;
    std::cout << "-------------testSum2Arrays-------------" << std::endl;
    std::cout << "--- mpi_rank = "             << mpi_rank             << std::endl;
    std::cout << "--- mpi_size = "             << mpi_size             << std::endl;
    std::cout << "--- cpuThreadsPerNode = "    << cpuThreadsPerNode    << std::endl;
    std::cout << "--- numElementsPerThread = " << numElementsPerThread << std::endl;
    std::cout << "--- nGpu = "                 << numGpu               << std::endl;
    std::cout << "--- numElementsPerGpu = "    << numElementsPerGpu    << std::endl;
    std::cout << "----------------------------------------" << std::endl;

    size_t numElementsInNode = cpuThreadsPerNode * numElementsPerThread + numGpu * numElementsPerGpu;
    size_t numElements = mpi_size * numElementsInNode;

    std::cout << "numElementsInNode = " << numElementsInNode    << std::endl;
    std::cout << "numElements = "       << numElements    << std::endl;

    //double* a = (double*)malloc(numElements * sizeof(*a));
    //double* b = (double*)malloc(numElements * sizeof(*b));
    //double* c = (double*)malloc(numElements * sizeof(*c));
    //double* c_par = (double*)malloc(numElements * sizeof(*c_par));
    
    double* a;
    double* b;
    double* c;
    double* c_par;
    hipHostAlloc((void**)&a, numElements * sizeof(*a), hipHostMallocDefault);
    hipHostAlloc((void**)&b, numElements * sizeof(*b), hipHostMallocDefault);
    hipHostAlloc((void**)&c, numElements * sizeof(*c), hipHostMallocDefault);
    hipHostAlloc((void**)&c_par, numElements * sizeof(*c_par), hipHostMallocDefault);
    

    for(int i = 0; i < numElements; i++)
    {
        a[i] = i;
        b[i] = 2.0 * i;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Последовательное суммирование
    hipEventRecord(start, 0);
    for(int i = 0; i < numElements; i++)
    {
        c[i] = a[i] + b[i];        
    }
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop);
    float elapsedTimeSeq;
    hipEventElapsedTime(&elapsedTimeSeq, start, stop);
    printf("Time of sequential summation: %lf sec\n", elapsedTimeSeq/1000);

    // Параллельное суммирование
    hipEventRecord(start, 0);
    sum2Arrays(a, b, c_par, cpuThreadsPerNode, numElementsPerThread,
        numGpu, numElementsPerGpu);
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop);    
    float elapsedTimePar;
    hipEventElapsedTime(&elapsedTimePar, start, stop);
    printf("Time of parallel summation: %lf sec\n", elapsedTimePar/1000);

    // Вывод первых 100 элементов массивов
    for(int i = 0; i < 100; i++)
    {
        std::cerr << "a[" << i << "] = " << a[i] << "; b[" << i << "] = " << b[i] << "; c[" << i << "] = " << c[i] << "; c_par[" << i << "] = " << c_par[i] << std::endl;
    }

    // Вывод неверно рассчитанных элементов массивов
    for(int i = 0; i < numElements; i++)
    {
        if(c[i]-c_par[i] > 0.001)
        {
            std::cerr << "ERROR! Checking stopped! " << "a[" << i << "] = " << a[i] << "; b[" << i << "] = " << b[i] << "; c[" << i << "] = " << c[i] << "; c_par[" << i << "] = " << c_par[i] << std::endl;
            break;
        }
    }

}
