#include "hip/hip_runtime.h"
#include <stdio.h>

//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

#define N 1000

using namespace std;

//
// A function marked __global__
// runs on the GPU but can be called from
// the CPU.
//
// This function multiplies the elements of an array
// of ints by 2.
//
// The entire computation can be thought of as running
// with one thread per array element with blockIdx.x
// identifying the thread.
//
// The comparison i<N is because often it isn't convenient
// to have an exact 1-1 correspondence between threads
// and array elements. Not strictly necessary here.
//
// Note how we're mixing GPU and CPU code in the same source
// file. An alternative way to use CUDA is to keep
// C/C++ code separate from CUDA code and dynamically
// compile and load the CUDA code at runtime, a little
// like how you compile and load OpenGL shaders from
// C/C++ code.
//
__global__
void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}


void DisplayHeader()
{
    const int kb = 1024;
    const int mb = kb * kb;
    wcout << "NBody.GPU" << endl << "=========" << endl << endl;

    wcout << "CUDA version:   v" << CUDART_VERSION << endl;    
    wcout << "Thrust version: v" << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION << endl << endl; 

    int devCount;
    hipGetDeviceCount(&devCount);
    wcout << "CUDA Devices: " << endl << endl;

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
        wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
        wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
        wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
        wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

        wcout << "  Warp size:         " << props.warpSize << endl;
        wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
        wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << endl;
        wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << endl;
        wcout << endl;
    }
}


int main() {
    DisplayHeader();

    //
    // Create int arrays on the CPU.
    // ('h' stands for "host".)
    //
    int ha[N], hb[N];

    //
    // Create corresponding int arrays on the GPU.
    // ('d' stands for "device".)
    //
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    //
    // Initialise the input data on the CPU.
    //
    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }

    //
    // Copy input data to array on GPU.
    //
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

    //
    // Launch GPU code with N threads, one per
    // array element.
    //
    add<<<N, 1>>>(da, db);

    

    //
    // Copy output array from GPU back to CPU.
    //
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }

    //
    // Free up the arrays on the GPU.
    //
    hipFree(da);
    hipFree(db);

    return 0;
}