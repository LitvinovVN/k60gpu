#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <sstream> // std::ostringstream

#include "gpu.h"


extern "C"
void printGpuParameters(std::string prefixDescr) {	
	int deviceCount;
	hipDeviceProp_t devProp;

	hipGetDeviceCount(&deviceCount);

	
	std::ostringstream ss;
	ss << "------- Printing CUDA-compatible device properties -------" << std::endl;
    ss << prefixDescr << std::endl;
	ss << "Finded " << deviceCount << " cuda compatible devices" << std::endl;

    std::cout << ss.str();
}


__global__ void mult(int x, int y, int *res) {	
	*res = x * y;	
}

extern "C"
int gpu(int x, int y){
	int *dev_res;	
	int res = 0;	
	hipMalloc((void**)&dev_res, sizeof(int));	
	mult<<<1,1>>>(x, y, dev_res);	
	hipMemcpy(&res, dev_res, sizeof(int), hipMemcpyDeviceToHost);	
	hipFree(dev_res);
	
	return res;
}