#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <sstream> // std::ostringstream

#include "gpu.h"


extern "C"
void printGpuParameters(std::string prefixDescr) {	
	int deviceCount;
	hipDeviceProp_t devProp;

	hipGetDeviceCount(&deviceCount);

	
	std::ostringstream ss;
	ss << "------- Printing CUDA-compatible device properties -------" << std::endl;
    ss << prefixDescr << std::endl;
	ss << "Finded " << deviceCount << " cuda compatible devices" << std::endl;

	for(int device = 0; device < deviceCount; device++){
		hipGetDeviceProperties(&devProp, device);

		ss << "Device " << device << std::endl;
		ss << "Compute capability: " << devProp.major << "." << devProp.minor << std::endl;
		ss << "Name   : " << devProp.name << std::endl;
	}

    std::cout << ss.str();
}


__global__ void mult(int x, int y, int *res) {	
	*res = x * y;	
}

extern "C"
int gpu(int x, int y){
	int *dev_res;	
	int res = 0;	
	hipMalloc((void**)&dev_res, sizeof(int));	
	mult<<<1,1>>>(x, y, dev_res);	
	hipMemcpy(&res, dev_res, sizeof(int), hipMemcpyDeviceToHost);	
	hipFree(dev_res);
	
	return res;
}