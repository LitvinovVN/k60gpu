#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#include "gpu.h"


extern "C"
void printGpuParameters(std::string prefixDescr) {	
	int deviceCount;
	hipDeviceProp_t devProp;

	hipGetDeviceCount(&deviceCount);

	std::cerr << prefixDescr << " printGpuParameters(): " << "deviceCount = " << deviceCount << std::endl;
}


__global__ void mult(int x, int y, int *res) {	
	*res = x * y;	
}

extern "C"
int gpu(int x, int y){
	int *dev_res;	
	int res = 0;	
	hipMalloc((void**)&dev_res, sizeof(int));	
	mult<<<1,1>>>(x, y, dev_res);	
	hipMemcpy(&res, dev_res, sizeof(int), hipMemcpyDeviceToHost);	
	hipFree(dev_res);
	
	return res;
}